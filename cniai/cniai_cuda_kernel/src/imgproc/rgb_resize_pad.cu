#include "hip/hip_runtime.h"
#include "cniai_cuda_kernel/common.h"
#include "cniai_cuda_kernel/imgproc.h"

namespace cniai_cuda_kernel::imgproc {

__global__ void rgbResizeBilinearPadKernel(const uint8_t *src, uint8_t *dst, int srcWidth,
                                           int srcHeight, int imgWidth, int imgHeight,
                                           int dstWidth, int dstHeight, int imgX,
                                           int imgY, int pad0, int pad1, int pad2,
                                           float scaleX, float scaleY,
                                           bool isOutputPlanar) {
    const unsigned int dstX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int dstY = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = 3;

    if (dstX >= dstWidth || dstY >= dstHeight)
        return;

    float srcX = static_cast<float>(dstX - imgX) * scaleX;
    float srcY = static_cast<float>(dstY - imgY) * scaleY;

    bool isInImg =
        imgY <= dstY && dstY < imgY + imgHeight && imgX <= dstX && dstX < imgX + imgWidth;
    for (int cIdx = 0; cIdx < channel; cIdx++) {
        float out = 0;
        if (isInImg) {
            const int x1 = __float2int_rd(srcX);
            const int y1 = __float2int_rd(srcY);
            const int x2 = x1 + 1;
            const int y2 = y1 + 1;
            const int x2Read = min(x2, srcWidth - 1);
            const int y2Read = min(y2, srcHeight - 1);

            uint8_t srcReg = src[y1 * srcWidth * channel + x1 * channel + cIdx];
            out = out + srcReg * ((x2 - srcX) * (y2 - srcY));

            srcReg = src[y1 * srcWidth * channel + x2Read * channel + cIdx];
            out = out + srcReg * ((srcX - x1) * (y2 - srcY));

            srcReg = src[y2Read * srcWidth * channel + x1 * channel + cIdx];
            out = out + srcReg * ((x2 - srcX) * (srcY - y1));

            srcReg = src[y2Read * srcWidth * channel + x2Read * channel + cIdx];
            out = out + srcReg * ((srcX - x1) * (srcY - y1));
        } else {
            out = cIdx == 0 ? pad0 : cIdx == 1 ? pad1 : pad2;
        }

        int dstCurrentIdx = isOutputPlanar
                                ? dstWidth * dstHeight * cIdx + dstY * dstWidth + dstX
                                : dstY * dstWidth * channel + dstX * channel + cIdx;

        dst[dstCurrentIdx] = out;
    }
}

void rgbResizeBilinearPad(const uint8_t *src, uint8_t *dst, int srcWidth, int srcHeight,
                          int imgWidth, int imgHeight, int dstWidth, int dstHeight,
                          int imgX, int imgY, int pad0, int pad1, int pad2,
                          bool isOutputPlanar, hipStream_t cudaStream) {
    dim3 block(32, 32);
    dim3 grid((dstWidth + block.x - 1) / block.x, (dstHeight + block.y - 1) / block.y);
    float scaleX = static_cast<float>(srcWidth) / static_cast<float>(imgWidth);
    float scaleY = static_cast<float>(srcHeight) / static_cast<float>(imgHeight);

    rgbResizeBilinearPadKernel<<<grid, block, 0, cudaStream>>>(
        src, dst, srcWidth, srcHeight, imgWidth, imgHeight, dstWidth, dstHeight, imgX,
        imgY, pad0, pad1, pad2, scaleX, scaleY, isOutputPlanar);
}

} // namespace cniai_cuda_kernel::imgproc